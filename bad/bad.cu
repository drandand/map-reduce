
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <iostream>
#include <regex>
#include <sstream>

void show_help(const std::string &cli)
{
    std::cout << "This explores what happens when copying memory to or from a CUDA device." << std::endl;
    std::cout << "If the sizes aren't matched, it's likely bad things happens, but it's not" << std::endl;
    std::cout << "what those bad things are.  This explores the badness." << std::endl;
    std::cout << std::endl;
    std::cout << "usage: " << cli << " <h_size> <d_size> <h_to_d_size> <d_to_h_size>" << std::endl;
    std::cout << std::endl;
    std::cout << "    h_size - number of ints to allocate on host." << std::endl;
    std::cout << "    d_size - number of ints to allocate on device." << std::endl;
    std::cout << "    h_to_d_size - number of ints to copy from host to device." << std::endl;
    std::cout << "    d_to_h_size - number of ints to copy from device to host." << std::endl;
    std::cout << std::endl;
}

std::string to_string(std::size_t size, int *arr)
{
    std::stringstream ss;

    char delim = '[';
    for (auto ptr = arr; ptr != arr + size; ++ptr)
    {
        ss << delim << (*ptr);
        delim = ',';
    }

    ss << ']';

    return ss.str();
}

int main(int argc, char *argv[])
{
    std::string cli = argv[0];
    std::regex pattern("^\\d+$");

    if (argc != 5)
    {
        show_help(cli);
        return -1;
    }

    std::vector<std::string> args(argv + 1, argv + argc);
    std::vector<std::size_t> vals;

    for (auto arg : args)
    {
        if (!std::regex_match(arg, pattern))
        {
            show_help(cli);
            return -1;
        }
        vals.push_back(std::stoi(arg));
    }

    const std::size_t h_size = vals[0];
    const std::size_t d_size = vals[1];
    const std::size_t h_to_d_size = vals[2];
    const std::size_t d_to_h_size = vals[3];

    std::cout << "assigning element on stack and allocating host array of " << h_size << " ints..." << std::endl;
    int h_fixed = 100;
    int *h_vec = new int[h_size];
    std::cout << "success\ninitializing host array..." << std::endl;
    for (std::size_t i = 0; i < h_size; ++i)
    {
        h_vec[i] = (i + 1) * (i + 2);
    }
    std::cout << "h_vec: " << std::hex << ((std::size_t)h_vec) << std::dec << " / " << to_string(h_size, h_vec) << std::endl;

    int *d_vec;
    std::cout << "success\nallocating device array of " << d_size << " ints." << std::endl;
    hipMalloc(&d_vec, d_size);

    std::cout << "success\ncopying " << h_to_d_size << " ints from host to device..." << std::endl;
    hipMemcpy(d_vec, h_vec, h_to_d_size * sizeof(int), hipMemcpyHostToDevice);

    std::cout << "success\ncopying " << d_to_h_size << " ints from device to host..." << std::endl;
    hipMemcpy(h_vec, d_vec, d_to_h_size * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "success" << std::endl;

    std::cout << "success\ncopying " << d_to_h_size << " ints from device to host (stack)..." << std::endl;
    hipMemcpy(&h_fixed, d_vec, d_to_h_size * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "success\n"
              << "h_vec: " << std::hex << ((std::size_t)h_vec) << std::dec << " / " << to_string(h_size, h_vec) << std::endl;
}