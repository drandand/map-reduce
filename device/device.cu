#include <map>
#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include "cuda_exception.cuh"
#include "cuda_helpers.cuh"
#include "cuda_constants.cuh"

/// @brief Show the information for all of the devices accessible to this host
void showCudaDeviceInfo()
{
    hipError_t err_code = hipInit(0);
    verify_code(err_code, "hipInit");

    int deviceCount;

    err_code = hipGetDeviceCount(&deviceCount);
    verify_code(err_code, "hipGetDeviceCount");

    for (std::size_t dev = 0; dev < deviceCount; ++dev)
    {
        char devName[256];
        hipDevice_t device;

        err_code = hipDeviceGet(&device, dev);
        verify_code(err_code, "hipDeviceGet");

        err_code = hipDeviceGetName(devName, 256, dev);
        verify_code(err_code, "hipDeviceGetName");
        std::cout << "device 0: " << std::string(devName) << std::endl;

        for (auto attr : DEVICE_ATTRIBUTE_VALUES)
        {
            int pi;
            std::string attr_name = DEVICE_ATTRIBUTE_NAMES.at(attr);

            err_code = hipDeviceGetAttribute(&pi, attr, device);
            verify_code(err_code, std::string("hipDeviceGetAttribute ") + attr_name);

            std::cout << "device " << dev << " / " << attr_name << ": " << pi << std::endl;
        }
    }
}

int main(int argc, char *argv[])
{
    try
    {
        showCudaDeviceInfo();
    }
    catch (cuda_exception &ex)
    {
        std::cerr << ex << std::endl;
    }
}